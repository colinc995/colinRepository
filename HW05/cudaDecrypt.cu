#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"


int main (int argc, char **argv) 
{

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */



  //declare storage for an ElGamal cryptosystem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */



  FILE *readPublic = fopen("public_key.txt", "r");

  fscanf(readPublic, "%u", &n);
  fscanf(readPublic, "%u", &p);
  fscanf(readPublic, "%u", &g);
  fscanf(readPublic, "%u", &h);

  fclose(readPublic);


  
  FILE *readMessage = fopen("message.txt", "r");

  fscanf(readMessage, "%u\n", &Nints);

  unsigned int *Zmessage = (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));

  for (int i=0; i < Nints; i++)
  {
    fscanf(readMessage, "%u %u\n", &Zmessage[i], &a[i]);
  } 

  fclose(readMessage);

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExp(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      } 
    }


    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }

  /* Q3 After finding the secret key, decrypt the message */

  ElGamalDecrypt(Zmessage, a, Nints, p, x);
  unsigned char *newArray = (unsigned char *) malloc(1024*sizeof(unsigned char));
  unsigned int Nchars = ((n-1)/8) * Nints;

  convertZToString(Zmessage, Nints, newArray, Nchars);
 


//4.1: search for parallel key

  int Nthreads;
  dim3 Bl(32,1,1);
  dim3 Gr((p+32-1)/32,1,1);
  unsigned int *device_a;
  unsigned int host_a;
  hipMalloc(&device_a, Nthreads*sizeof(unsigned int));
  host_a = (unsigned int *) malloc(sizeof(unsigned int));

  double start = clock();
  cudaDecrypt <<<Gr,Bl>>>(g,p,h,device_a);
  hipDeviceSynchronize();

  hipMemcpy(host_A,device_a,sizeof(unsigned int) , hipMemcpyDeviceToHost);



  return 0;


}










__device__ unsigned int cudaModExp(unsigned int a, unsigned int b, unsigned p)
{
   unsigned int z = a;
   unsigned int aExpb = 1;

   while (b > 0) {
     if (b%2 == 1) aExpb = cudaModprod(aExpb, z, p);
     z = cudaModprod(z, z, p);
     b /= 2;
   }
   return aExpb;
}




__device__ unsigned int cudaModProd(unsigned int a, unsigned int b, unsigned p)
{
    unsigned int z_a = a;
    unsigned int a_b = 0;

    while (b>0)
    {
        if (b%2 == ) a_b = (a_b + z_a)%p;
        z_a = (2*z_a) % p;
        b/= 2;
    }
    return a_b;
}







// find the key
__global__ void cudaKeyDecrypt(unsigned int g, unsigned int p, unsigned int h, unsigned int *d)
{
    int thread = threadIdx.x;
    int block = blockIdx.x;
    int blockSize = blockDim.x;
    int gid = thread + block*blockSize;

    if (gid < (p-1))
    {
        if (cudaModExp(g,gid,p) == h)
        {
            d = gid-1;
        }
    }
}




